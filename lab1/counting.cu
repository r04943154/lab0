#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>

#include <cstdio>
#include <cstdlib>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <cmath>

using namespace std;


__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

// determines whether the character is alphabetical
__host__ __device__
bool is_alpha(const char c)
{
	return (c >= 'a' && c <= 'z');
}

// determines whether the right character begins a new word
struct is_word_start
	: public thrust::binary_function<const char&, const char&, bool>
{
	__host__ __device__
		bool operator()(const char& left, const char& right) const
	{
		return is_alpha(right) && !is_alpha(left);
	}
};
int word_count(const thrust::device_vector<int>& input)
{
	// check for empty string
	if (input.empty())
		return 0;

	// compute the number characters that start a new word
	int wc = thrust::inner_product(input.begin(), input.end() - 1,  // sequence of left characters
		input.begin() + 1,               // sequence of right characters
		0,                               // initialize sum to 0
		thrust::plus<int>(),             // sum values together
		is_word_start());       // how to compare the left and right characters

	// if the first character is alphabetical, then it also begins a word
	if (is_alpha(input.front()))
		wc++;

	return wc;
}

__global__ void toOne(const char* A, int* B, int C)
{
	int i =  blockIdx.x + threadIdx.x;
	if (A[i] != '\n'){
		//printf(".@%d.",i);
		B[i] = 1;
	}
	else
	{
		B[i] = 0; 
		//printf(".~%d.", i);
	}
	/*int j = 1;
	for (int hi = 0; hi < C; hi++){
		if (A[hi] != '\n'){
			B[hi] = j;
			j++;
		}
		else{
			B[hi] = 0;
			j = 1;
		}
	}*/
}
__global__ void STadd(int A, int* B, int C)
{
	int i = blockIdx.x + threadIdx.x;
	if (B[i*2]==1&&B[i*2+1]==1)
	{
		B[C + i] = 2;
		if (i > 0){
			if (B[i * 2 - 2] == 0 && B[i * 2 - 1] == A && B[i * 2 + 2] == 0 && B[i * 2 + 3] == 0)B[C + i] = 2+A;
			if (B[i * 2 - 2] == 0 && B[i * 2 - 1] == A && B[i * 2 + 2] == A && B[i * 2 + 3] == 0)B[C + i] = 2+A+A;
			if (B[i * 2 - 2] == 0 && B[i * 2 - 1] == 0 && B[i * 2 + 2] == A && B[i * 2 + 3] == 0)B[C + i] = 2+A;
		}
	}
}

void CountPosition(const char *text, int *pos, int text_size)
{
	cout << endl << pos << endl;

	int numofblock = text_size / 256 + 1;
	toOne << <1, 1 >> >(text, pos, text_size);
	for (int i = 1; i < 11; i++){
		hipDeviceSynchronize();
		STadd << <numofblock/pow(2,i), 256 >> >( i, pos, text_size);
	}
	int j = 1;


	thrust::device_ptr<const char> text_d(text);
	thrust::device_vector<char> input(text_d, text_d + 1000);

	// count words
	//int wc = word_count(input);

	//std::cout << "Text sample contains " << wc << " words" << std::endl;
	
}




int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	int nhead=1;
	hipMalloc(&buffer, sizeof(int)*text_size*2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer+text_size);

	// TODO
	cout << endl << pos << endl;
	//thrust::inclusive_scan(thrust::host, pos_d, pos_d + text_size, head_d);
	
	hipFree(buffer);
	return nhead;
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
}
