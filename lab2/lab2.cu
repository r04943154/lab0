#include "hip/hip_runtime.h"
#include "timer.h"   

#include "lab2.h"

#include "hip/hip_runtime.h"
#include ""


static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 240;


__global__ void simple_kernel(uint8_t *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	
	float u = x / (float)width;
	float v = y / (float)height;
	u = u*2.0f - 1.0f;
	v = v*2.0f - 1.0f;

	
	float freq = 4.0f;
	float w = sinf(u*freq + time/30) * cosf(v*freq + time/30) * 2.5f;

	if (time>120)w = sinf(u*freq + time/60) * cosf(v*freq + time/60) * 40.5f;

	
	//pos[y*width + x] = make_float4(u, w, v, 1.0f);
	pos[y*width + x] = (uint8_t)(u + v + w )*1000 % 256;
	if (time<40||time>200)pos[y*width + x] = (uint8_t)(x+y+time) % 256;
	//printf("=%f %f %f=\n",u , v, w);
}


struct Lab2VideoGenerator::Impl {
	int t = 0;
};

Lab2VideoGenerator::Lab2VideoGenerator(): impl(new Impl) {
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};


void Lab2VideoGenerator::Generate(uint8_t *yuv) {

	
	//float t = (float)time();

	dim3 block(8, 8, 1);
	dim3 grid(W / block.x, H*1.5 / block.y, 1);
	simple_kernel << < grid, block >> >(yuv, W, H, impl->t);

	//hipMemset(yuv, (impl->t), W*H/2);
	//hipMemset(yuv + W*H / 2, ((impl->t)+128)%256, W*H/2);
	//hipMemset(yuv + W*H, rand(), W*H / 2);
	//impl->t = rand() % 256;
	impl->t += 1;
}
