#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "SyncedMemory.h"
#include ""
#include<stdlib.h>
#include<stdio.h>
#include<iostream>
#include <fstream>

using namespace std;



__global__ void SomeTransform(char *input_gpu, int fsize) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < fsize && input_gpu[idx] != '\n') {
		input_gpu[idx] = input_gpu[idx]+2;//let every char ASCII code plus two
	}
}

int main()
{
	// init, and check
	char file[] = "test.txt";
	
	FILE *fp = fopen("test.txt", "r");
	if (!fp) {
		printf("Cannot open %s", file);
		//abort();
	}
	// get file size
	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	// read files
	MemoryBuffer<char> text(fsize + 1);
	auto text_smem = text.CreateSync(fsize);

	fread(text_smem.get_cpu_wo(), 1, fsize, fp);
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);

	// TODO: do your transform here
	char *input_gpu = text_smem.get_gpu_rw();
	// An example: transform the first 64 characters to '!'
	// Don't transform over the tail
	// And don't transform the line breaks
	SomeTransform << <2, fsize >> >(input_gpu, fsize);

	puts(text_smem.get_cpu_ro());
	printf("%d" ,text_smem );
	system("pause");
	return 0;
}
