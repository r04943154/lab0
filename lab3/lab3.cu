#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht && xt < wt && mask[curt] > 127.0f) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if (0 <= yb && yb < hb && 0 <= xb && xb < wb) {
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
		}
	}
}

__global__ void CalculateFixed(const float *background, const float *target, const float *mask, float *fixed,
	                           const int wb, const int hb, const int wt, const int ht, const int oy, const int ox){
	float nb[3] = { 0.0, 0.0, 0.0 }, sb[3] = { 0.0, 0.0, 0.0 }, eb[3] = { 0.0, 0.0, 0.0 }, wwb[3] = { 0.0, 0.0, 0.0 };
	float nt[3] = { 0.0, 0.0, 0.0 }, st[3] = { 0.0, 0.0, 0.0 }, et[3] = { 0.0, 0.0, 0.0 }, wwt[3] = { 0.0, 0.0, 0.0 };
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt + xt;
	const int yb = oy + yt, xb = ox + xt;
	const int curb = wb*yb + xb;

	if (mask[curt] == 0){
		//fixed[curt * 3 + 0] = background[curb * 3 + 0];
		//fixed[curt * 3 + 1] = background[curb * 3 + 1];
		//fixed[curt * 3 + 2] = background[curb * 3 + 2];
	}
	else{
		/////N
		if (curt >= wt){
			if (mask[curt - wt] == 255){
				//nb[0] = background[(curb - wb) * 3 + 0];
				//nb[1] = background[(curb - wb) * 3 + 1];
				//nb[2] = background[(curb - wb) * 3 + 2];
			}
			else{
				nb[0] = background[(curb - wb) * 3 + 0];
				nb[1] = background[(curb - wb) * 3 + 1];
				nb[2] = background[(curb - wb) * 3 + 2];
				//nb[0] = fixed[(curt - wt) * 3 + 0];
				//nb[1] = fixed[(curt - wt) * 3 + 1];
				//nb[2] = fixed[(curt - wt) * 3 + 2];
				//mask[curt] = 0;
			}
			nt[0] = target[(curt - wt) * 3 + 0];
			nt[1] = target[(curt - wt) * 3 + 1];
			nt[2] = target[(curt - wt) * 3 + 2];
		}
		else{
			nt[0] = target[curt * 3 + 0];
			nt[1] = target[curt * 3 + 1];
			nt[2] = target[curt * 3 + 2];
			nb[0] = background[(curb - wb) * 3 + 0];
			nb[1] = background[(curb - wb) * 3 + 1];
			nb[2] = background[(curb - wb) * 3 + 2];
		}
		///////////////////////S
		if (curt + wt<wt*ht){
			if (mask[curt + wt] == 255){
				//sb[0] = background[(curb + wb) * 3 + 0];
				//sb[1] = background[(curb + wb) * 3 + 1];
				//sb[2] = background[(curb + wb) * 3 + 2];
			}
			else{
				sb[0] = background[(curb + wb) * 3 + 0];
				sb[1] = background[(curb + wb) * 3 + 1];
				sb[2] = background[(curb + wb) * 3 + 2];
				//sb[0] = fixed[(curt + wt) * 3 + 0];
				//sb[1] = fixed[(curt + wt) * 3 + 1];
				//sb[2] = fixed[(curt + wt) * 3 + 2];
			}
			st[0] = target[(curt + wt) * 3 + 0];
			st[1] = target[(curt + wt) * 3 + 1];
			st[2] = target[(curt + wt) * 3 + 2];
		}
		else{
			st[0] = target[curt * 3 + 0];
			st[1] = target[curt * 3 + 1];
			st[2] = target[curt * 3 + 2];
			sb[0] = background[(curb + wb) * 3 + 0];
			sb[1] = background[(curb + wb) * 3 + 1];
			sb[2] = background[(curb + wb) * 3 + 2];
			//sb[0] = fixed[curt * 3 + 0];
			//sb[1] = fixed[curt * 3 + 1];
			//sb[2] = fixed[curt * 3 + 2];
		}
		///////////////////W
		if (curt%wt != 0){
			if (mask[curt - 1] == 255){
				//wwb[0] = background[(curb - 1) * 3 + 0];
				//wwb[1] = background[(curb - 1) * 3 + 1];
				//wwb[2] = background[(curb - 1) * 3 + 2];
			}
			else{
				wwb[0] = background[(curb - 1) * 3 + 0];
				wwb[1] = background[(curb - 1) * 3 + 1];
				wwb[2] = background[(curb - 1) * 3 + 2];
				//wwb[0] = fixed[(curt - 1) * 3 + 0];
				//wwb[1] = fixed[(curt - 1) * 3 + 1];
				//wwb[2] = fixed[(curt - 1) * 3 + 2];
			}
			wwt[0] = target[(curt - 1) * 3 + 0];
			wwt[1] = target[(curt - 1) * 3 + 1];
			wwt[2] = target[(curt - 1) * 3 + 2];
		}
		else{
			wwt[0] = target[curt * 3 + 0];
			wwt[1] = target[curt * 3 + 1];
			wwt[2] = target[curt * 3 + 2];
			wwb[0] = background[(curb - 1) * 3 + 0];
			wwb[1] = background[(curb - 1) * 3 + 1];
			wwb[2] = background[(curb - 1) * 3 + 2];
			//wwb[0] = fixed[curt * 3 + 0];
			//wwb[1] = fixed[curt * 3 + 1];
			//wwb[2] = fixed[curt * 3 + 2];
		}
		///////////////E
		if ((curt + 1) % wt != 0){
			if (mask[curt + 1] == 255){
				//eb[0] = background[(curb + 1) * 3 + 0];
				//eb[1] = background[(curb + 1) * 3 + 1];
				//eb[2] = background[(curb + 1) * 3 + 2];
			}
			else{
				eb[0] = background[(curb + 1) * 3 + 0];
				eb[1] = background[(curb + 1) * 3 + 1];
				eb[2] = background[(curb + 1) * 3 + 2];
				//eb[0] = fixed[(curt + 1) * 3 + 0];
				//eb[1] = fixed[(curt + 1) * 3 + 1];
				//eb[2] = fixed[(curt + 1) * 3 + 2];
			}
			et[0] = target[(curt + 1) * 3 + 0];
			et[1] = target[(curt + 1) * 3 + 1];
			et[2] = target[(curt + 1) * 3 + 2];
		}
		else{
			et[0] = target[curt * 3 + 0];
			et[1] = target[curt * 3 + 1];
			et[2] = target[curt * 3 + 2];
			eb[0] = background[(curb + 1) * 3 + 0];
			eb[1] = background[(curb + 1) * 3 + 1];
			eb[2] = background[(curb + 1) * 3 + 2];
			//eb[0] = fixed[curt * 3 + 0];
			//eb[1] = fixed[curt * 3 + 1];
			//eb[2] = fixed[curt * 3 + 2];
		}
		fixed[curt * 3 + 0] = 4 * target[curt * 3 + 0] - nt[0] - st[0] - wwt[0] - et[0] + nb[0] + sb[0] + wwb[0] + eb[0];
		fixed[curt * 3 + 1] = 4 * target[curt * 3 + 1] - nt[1] - st[1] - wwt[1] - et[1] + nb[1] + sb[1] + wwb[1] + eb[1];
		fixed[curt * 3 + 2] = 4 * target[curt * 3 + 2] - nt[2] - st[2] - wwt[2] - et[2] + nb[2] + sb[2] + wwb[2] + eb[2];
	}
}


__global__ void PoissonImageCloneing(float *fixed, const float *mask, const float *buf1, float *buf2, const int wt, const int ht)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt + xt;
	//float nb[3] = { 255.0, 255.0, 255.0 }, sb[3] = { 255.0, 255.0, 255.0 }, eb[3] = { 255.0, 255.0, 255.0 }, wwb[3] = { 255.0, 255.0, 255.0 };
	//float nt[3] = { 255.0, 255.0, 255.0 }, st[3] = { 255.0, 255.0, 255.0 }, et[3] = { 255.0, 255.0, 255.0 }, wwt[3] = { 255.0, 255.0, 255.0 };
	float nb[3] = { 0.0, 0.0, 0.0 }, sb[3] = { 0.0, 0.0, 0.0 }, eb[3] = { 0.0, 0.0, 0.0 }, wwb[3] = { 0.0, 0.0, 0.0 };
	float nt[3] = { 0.0, 0.0, 0.0 }, st[3] = { 0.0, 0.0, 0.0 }, et[3] = { 0.0, 0.0, 0.0 }, wwt[3] = { 0.0, 0.0, 0.0 };

	/////N
	if (curt >= wt){
		if (mask[curt - wt] == 255){
			nb[0] = buf1[(curt - wt) * 3 + 0];
			nb[1] = buf1[(curt - wt) * 3 + 1];
			nb[2] = buf1[(curt - wt) * 3 + 2];
		}
		else{
			//nb[0] = fixed[(curt - wt) * 3 + 0];
			//nb[1] = fixed[(curt - wt) * 3 + 1];
			//nb[2] = fixed[(curt - wt) * 3 + 2];
			//mask[curt] = 0;
		}		
	}
	else{
		//nb[0] = buf1[curt * 3 + 0];
		//nb[1] = buf1[curt * 3 + 1];
		//nb[2] = buf1[curt * 3 + 2];
	}
	///////////////////////S
	if (curt + wt<wt*ht){
		if (mask[curt + wt] == 255){
			sb[0] = buf1[(curt + wt) * 3 + 0];
			sb[1] = buf1[(curt + wt) * 3 + 1];
			sb[2] = buf1[(curt + wt) * 3 + 2];
		}
		else{

		}
		
	}
	else{
	
		//sb[0] = buf1[curt * 3 + 0];
		//sb[1] = buf1[curt * 3 + 1];
		//sb[2] = buf1[curt * 3 + 2];
	}
	///////////////////W
	if (curt%wt != 0){
		if (mask[curt - 1] == 255){
			wwb[0] = buf1[(curt - 1) * 3 + 0];
			wwb[1] = buf1[(curt - 1) * 3 + 1];
			wwb[2] = buf1[(curt - 1) * 3 + 2];
		}
		else{
		
		}
	
	}
	else{
	
		//wwb[0] = buf1[curt * 3 + 0];
		//wwb[1] = buf1[curt * 3 + 1];
		//wwb[2] = buf1[curt * 3 + 2];
	}
	///////////////E
	if ((curt + 1) % wt != 0){
		if (mask[curt + 1] == 255){
			eb[0] = buf1[(curt + 1) * 3 + 0];
			eb[1] = buf1[(curt + 1) * 3 + 1];
			eb[2] = buf1[(curt + 1) * 3 + 2];
		}
		else{
			
		}
		
	}
	else{
		
		//eb[0] = buf1[curt * 3 + 0];
		//eb[1] = buf1[curt * 3 + 1];
		//eb[2] = buf1[curt * 3 + 2];
	}
	if (mask[curt] == 255){
		buf2[curt * 3 + 0] = (fixed[curt * 3 + 0] + nb[0] + sb[0] + wwb[0] + eb[0]) / 4;
		buf2[curt * 3 + 1] = (fixed[curt * 3 + 1] + nb[1] + sb[1] + wwb[1] + eb[1]) / 4;
		buf2[curt * 3 + 2] = (fixed[curt * 3 + 2] + nb[2] + sb[2] + wwb[2] + eb[2]) / 4;
	}
	/*
	/////N
	if (curt>=wt){
		if (mask[curt - wt]==255){
			nb[0] = buf1[(curt - wt) * 3 + 0];
			nb[1] = buf1[(curt - wt) * 3 + 1];
			nb[2] = buf1[(curt - wt) * 3 + 2];
		}
		else{
			nb[0] = fixed[(curt - wt) * 3 + 0];
			nb[1] = fixed[(curt - wt) * 3 + 1];
			nb[2] = fixed[(curt - wt) * 3 + 2];
			//mask[curt] = 0;
		}
		nt[0] = fixed[(curt - wt) * 3 + 0];
		nt[1] = fixed[(curt - wt) * 3 + 1];
		nt[2] = fixed[(curt - wt) * 3 + 2];
	}
	else{
		nt[0] = fixed[curt * 3 + 0];
		nt[1] = fixed[curt * 3 + 1];
		nt[2] = fixed[curt * 3 + 2];
		nb[0] = fixed[curt * 3 + 0];
		nb[1] = fixed[curt * 3 + 1];
		nb[2] = fixed[curt * 3 + 2];
	}
	///////////////////////S
	if (curt + wt<wt*ht){
		if (mask[curt + wt]==255){
			sb[0] = buf1[(curt + wt) * 3 + 0];
			sb[1] = buf1[(curt + wt) * 3 + 1];
			sb[2] = buf1[(curt + wt) * 3 + 2];
		}
		else{
			sb[0] = fixed[(curt + wt) * 3 + 0];
			sb[1] = fixed[(curt + wt) * 3 + 1];
			sb[2] = fixed[(curt + wt) * 3 + 2];
		}
		st[0] = fixed[(curt + wt) * 3 + 0];
		st[1] = fixed[(curt + wt) * 3 + 1];
		st[2] = fixed[(curt + wt) * 3 + 2];
	}
	else{
		st[0] = fixed[curt * 3 + 0];
		st[1] = fixed[curt * 3 + 1];
		st[2] = fixed[curt * 3 + 2];
		sb[0] = fixed[curt * 3 + 0];
		sb[1] = fixed[curt * 3 + 1];
		sb[2] = fixed[curt * 3 + 2];
	}
	///////////////////W
	if (curt%wt != 0){
		if (mask[curt - 1]==255){
			wwb[0] = buf1[(curt - 1) * 3 + 0];
			wwb[1] = buf1[(curt - 1) * 3 + 1];
			wwb[2] = buf1[(curt - 1) * 3 + 2];
		}
		else{
			wwb[0] = fixed[(curt - 1) * 3 + 0];
			wwb[1] = fixed[(curt - 1) * 3 + 1];
			wwb[2] = fixed[(curt - 1) * 3 + 2];
		}
		wwt[0] = fixed[(curt - 1) * 3 + 0];
		wwt[1] = fixed[(curt - 1) * 3 + 1];
		wwt[2] = fixed[(curt - 1) * 3 + 2];
	}
	else{
		wwt[0] = fixed[curt * 3 + 0];
		wwt[1] = fixed[curt * 3 + 1];
		wwt[2] = fixed[curt * 3 + 2];
		wwb[0] = fixed[curt * 3 + 0];
		wwb[1] = fixed[curt * 3 + 1];
		wwb[2] = fixed[curt * 3 + 2];
	}
	///////////////E
	if ((curt + 1) % wt != 0){
		if (mask[curt + 1]==255){
			eb[0] = buf1[(curt + 1) * 3 + 0];
			eb[1] = buf1[(curt + 1) * 3 + 1];
			eb[2] = buf1[(curt + 1) * 3 + 2];
		}
		else{
			eb[0] = fixed[(curt + 1) * 3 + 0];
			eb[1] = fixed[(curt + 1) * 3 + 1];
			eb[2] = fixed[(curt + 1) * 3 + 2];
		}
		et[0] = fixed[(curt + 1) * 3 + 0];
		et[1] = fixed[(curt + 1) * 3 + 1];
		et[2] = fixed[(curt + 1) * 3 + 2];
	}
	else{
		et[0] = fixed[curt * 3 + 0];
		et[1] = fixed[curt * 3 + 1];
		et[2] = fixed[curt * 3 + 2];
		eb[0] = fixed[curt * 3 + 0];
		eb[1] = fixed[curt * 3 + 1];
		eb[2] = fixed[curt * 3 + 2];
	}
	if (mask[curt] == 255){
		buf2[curt * 3 + 0] = (4 * buf1[curt * 3 + 0] - nt[0] - st[0] - wwt[0] - et[0] + nb[0] + sb[0] + wwb[0] + eb[0]) / 4;
		buf2[curt * 3 + 1] = (4 * buf1[curt * 3 + 1] - nt[1] - st[1] - wwt[1] - et[1] + nb[1] + sb[1] + wwb[1] + eb[1]) / 4;
		buf2[curt * 3 + 2] = (4 * buf1[curt * 3 + 2] - nt[2] - st[2] - wwt[2] - et[2] + nb[2] + sb[2] + wwb[2] + eb[2]) / 4;
	}
	*/
}

void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	float *fixed, *buf1, *buf2;
	hipMalloc(&fixed, 3 * wt*ht*sizeof(float));
	hipMalloc(&buf1, 3 * wt*ht*sizeof(float));
	hipMalloc(&buf2, 3 * wt*ht*sizeof(float));

	// intialize
	dim3 gdim(CeilDiv(wt, 32), CeilDiv(ht, 16)), bdim(32, 16);
	CalculateFixed<<<gdim, bdim>>>(background, target, mask, fixed, wb, hb, wt, ht, oy, ox);
	hipMemcpy(buf1, target, wt*ht*sizeof(float) * 3, hipMemcpyDeviceToDevice);

	//iterate
	for (int i = 0; i < 10000; i++){
		PoissonImageCloneing<<<gdim, bdim>>>(fixed, mask, buf1, buf2, wt, ht);
		PoissonImageCloneing<<<gdim, bdim>>>(fixed, mask, buf2, buf1, wt, ht);
	}


	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	SimpleClone<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
		background, buf1, mask, output,
		wb, hb, wt, ht, oy, ox
	);
	
}
